#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <iostream>
#include <string>

/* Include polybench common header. */
#include "polybench.hu"

/* Include benchmark-specific header. */
/* Default data type is double, default size is 4000. */
#include "atax.hu"

// Workaround for the editor not finding M_PI
// It is exclusive to the GNU C compiler
// https://www.gnu.org/software/libc/manual/html_node/Mathematical-Constants.html
#ifndef M_PI
	#define M_PI 3.141
#endif

// Default if CUDA_NTHREADS is not set
#ifndef CUDA_NTHREADS
	#define CUDA_NTHREADS 128
#endif


/**
 * Given a `x` and a `y`, compute the relative index of the element in the `A` matrix.
 */
__host__ __device__ inline static unsigned int a_index(unsigned int x, unsigned int y) {
	return x * NY + y;
}

/**
 * Log a debug message.
 */
__host__ inline static void print_debug(std::string txt) {
	#ifdef HPC_DEBUG
		std::cerr << txt << std::endl;
	#endif
}

/**
 * Log an error message.
 */
#ifdef HPC_USE_CUDA
__host__ inline static void print_cudaError(hipError_t err, std::string txt) {
	#ifdef HPC_DEBUG
		std::cerr << txt;
		fprintf( stderr, ": error in file '%s' in line %i: %s.\n", __FILE__, __LINE__, hipGetErrorString(err) );
	#endif
}
#endif

/**
 * Initialize the arrays to be used in the computation:
 * 
 * - `X` is filled with multiples of `M_PI`;
 * - `Y` is zeroed;
 * - `A` is filled with sample data.
 * 
 * To be called on the CPU (uses the `__host__` qualifier).
 */
#ifndef HPC_USE_CUDA
__host__ static void init_array(DATA_TYPE* A, DATA_TYPE* X, DATA_TYPE* Y)
{
	for (unsigned int y = 0; y < NY; y++) 
	{
		X[y] = y * M_PI;
	}

	for (unsigned int x = 0; x < NX; x++) 
	{
		Y[x] = 0;
	}

	for (unsigned int x = 0; x < NX; x++) 
	{
		for (unsigned int y = 0; y < NY; y++) 
		{
			A[a_index(x, y)] = (DATA_TYPE)(x * (y + 1)) / NX;
		}
	}
}
#endif

/**
 * Initialize the `X` array.
 * 
 * Runs on the device.
 */
#ifdef HPC_USE_CUDA
__device__ static void init_array_cuda_x(DATA_TYPE* X, unsigned int threads)
{
	// Find how many iterations should be performed by each thread
	unsigned int perThread = NY / threads + 1;

	// Find the index of the current thread, even if threads span multiple blocks
	int blockThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;
	
	// Have each thread perform the previously determined number of iterations
	for(int stride = 0; stride < perThread; stride++)
	{
		// Find the index of the current iteration
		// This is equal to `y` of the init_array function
		unsigned int iterationIdx = threads * stride + blockThreadIdx;

		// Prevent the thread from accessing unallocated memory
		if(iterationIdx < NY)
		{
			// Set the array element
			X[iterationIdx] = iterationIdx * M_PI;
		}
	}
}
#endif

/**
 * Initialize the `Y` array.
 * 
 * Runs on the device.
 */
#ifdef HPC_USE_CUDA
__device__ static void init_array_cuda_y(DATA_TYPE* Y, unsigned int threads)
{
	// Find how many iterations should be performed by each thread
	unsigned int perThread = NX / threads + 1;

	// Find the index of the current thread, even if threads span multiple blocks
	int blockThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;
	
	// Have each thread perform the previously determined number of iterations
	for(int stride = 0; stride < perThread; stride++) 
	{
		// Find the index of the current iteration
		// This is equal to `y` of the init_array function
		unsigned int iterationIdx = threads * stride + blockThreadIdx;

		// Prevent the thread from accessing unallocated memory
		if(iterationIdx < NX) 
		{
			// Set the array element
			Y[iterationIdx] = 0;
		}
	}
}
#endif

/**
 * Initialize the `A` array.
 * 
 * Runs on the device.
*/
#ifdef HPC_USE_CUDA
__device__ static void init_array_cuda_a(DATA_TYPE* A, unsigned int threads)
{
	// Find how many elements should be written in total
	unsigned int elements = NX * NY;

	// Find how many iterations should be performed by each thread
	unsigned int perThread = elements / threads + 1;

	// Find the index of the current thread, even if threads span multiple blocks
	int blockThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;

	// Have each thread perform the previously determined number of iterations
	for(int stride = 0; stride < perThread; stride++) 
	{
		// Find the index of the current iteration
		// This is equal to `y` of the init_array function
		unsigned int iterationIdx = threads * stride + blockThreadIdx;

		// Determine current x and y
		unsigned int y = iterationIdx % NY;
		unsigned int x = iterationIdx / NY;

		// Prevent the thread from accessing unallocated memory
		if(iterationIdx < elements) 
		{
			// Set the array element
			A[iterationIdx] = (DATA_TYPE)(x * (y + 1)) / NX;
		}
	}
}
#endif

/**
 * Initialize the arrays to be used in the computation:
 * 
 * - `X` is filled with multiples of `M_PI`;
 * - `Y` is zeroed;
 * - `A` is filled with sample data.
 * 
 * Beware that `A` here is a simple array, it is not a matrix, so elements are accessed via [y * NX + x] (I think?).
 * 
 * It is called by the host, runs on the device, and calls the other init_arrays on the device.
 */
#ifdef HPC_USE_CUDA
__global__ static void init_array_cuda(DATA_TYPE* A, DATA_TYPE* X, DATA_TYPE* Y)
{
	unsigned int threads = gridDim.x * blockDim.x;

	init_array_cuda_x(X, threads);
	init_array_cuda_y(Y, threads);
	init_array_cuda_a(A, threads);
}
#endif

/**
 * Print the given array.
 * 
 * Cannot be parallelized, as the elements of the array should be 
 * 
 * To be called on the CPU (uses the `__host__` qualifier).
 */
#ifdef HPC_DEBUG
__host__ static void print_array(DATA_TYPE* Z, unsigned int size)
{
	for (unsigned int z = 0; z < size; z++) 
	{
		fprintf(stderr, DATA_PRINTF_MODIFIER, Z[z]);
	}
	fprintf(stderr, "\n");
}
#endif


/**
 * Compute ATAX :
 * - A is the input matrix
 * - X is an input vector
 * - Y is the result vector
 * 
 * In particular:
 * ```
 * A * (A * X) = Y
 * ```
 * Wait, there's no transposition here?!?
 * 
 * Parallelizing this is the goal of the assignment.
 * 
 * To be called on the CPU uses the `__host__` qualifier otherwise
 * for the GPU uses the `__global__` qualifier.
 */
#ifndef HPC_USE_CUDA
__host__ static void kernel_atax(DATA_TYPE* A, DATA_TYPE* X, DATA_TYPE* Y)
{
	for (unsigned int x = 0; x < NY; x++) 
	{
		DATA_TYPE tmp = 0;
		
		for (unsigned int y = 0; y < NX; y++) 
		{
			tmp += A[a_index(x, y)] * X[y];
		}
		
		for (unsigned int y = 0; y < NX; y++) 
		{
			Y[x] += A[a_index(x, y)] * tmp;
		}
	}
}
#else

__global__ static void kernel_atax_cuda(DATA_TYPE* A, DATA_TYPE* X, DATA_TYPE* Y)
{
	// Find out how many threads there are
	unsigned int threads = gridDim.x * blockDim.x;

	// Find how many iterations should be performed by each thread
	unsigned int perThread = NX / threads + 1;

	// Find the index of the current thread, even if threads span multiple blocks
	unsigned int blockThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;

	// Have each thread perform the previously determined number of iterations
	for(int stride = 0; stride < perThread; stride++) 
	{
		// Iterate over x; y is not parallelized
		unsigned int x = threads * stride + blockThreadIdx;
		
		// Prevent the thread from accessing unallocated memory
		if(x < NX) 
		{
			// The same tmp as earlier
			DATA_TYPE tmp = 0;

			for (unsigned int y = 0; y < NX; y++) 
			{
				tmp += A[a_index(x, y)] * X[y];
			}

			for (unsigned int y = 0; y < NX; y++) 
			{
				// THIS DOES NOT WORK ON THE NANO, AS IT IS TOO OLD TO SUPPORT ATOMIC ADDITION WITH DOUBLES!
				// If you want to use the Nano, swap this for something else, or change atax.hu to use float instead of double
				atomicAdd(&Y[x], A[a_index(x, y)] * tmp);
			}
		}
	}
}
#endif


/**
 * The main function of the benchmark, which sets up tooling to measure the time spent computing `kernel_atax`.
 * 
 * We should probably avoid editing this.
 */
__host__ int main(int argc, char** argv)
{
	print_debug("[Main] Starting...");
	std::cerr << "[Main] NX is: " << NX << std::endl;
	std::cerr << "[Main] NY is: " << NY << std::endl;

	#ifndef HPC_USE_CUDA

		print_debug("[Mode] Host-only");

		print_debug("[Pointers] Allocating...");

		DATA_TYPE* A = new DATA_TYPE[NX * NY];
		DATA_TYPE* X = new DATA_TYPE[NY];
		volatile DATA_TYPE* Y = new DATA_TYPE[NX];

		print_debug("[Pointers] Allocated!");

		#ifdef HPC_INCLUDE_INIT
			print_debug("[Benchmark] Starting...");
			polybench_start_instruments;
		#endif

		print_debug("[Init] Initializing...");
		init_array(A, X, (DATA_TYPE*) Y);
		print_debug("[Init] Initialized!");

		#ifndef HPC_INCLUDE_INIT
			print_debug("[Benchmark] Starting...");
			polybench_start_instruments;
		#endif

		print_debug("[Kernel] Running...");
		kernel_atax(A, X, (DATA_TYPE*) Y);
		print_debug("[Kernel] Completed!");

		print_debug("[Benchmark] Stopping...");
		polybench_stop_instruments;
		polybench_print_instruments;
		print_debug("[Benchmark] Complete!");

		#ifdef HPC_DEBUG
			print_debug("[Debug] Displaying A:");
			print_array(A, NX * NY);
			print_debug("[Debug] Displaying X:");
			print_array(X, NY);
			print_debug("[Debug] Displaying Y:");
			print_array(Y, NX);
		#endif

	#else
	
		print_debug("[Mode] Host-and-device, CUDA");

		print_debug("[Pointers] Allocating...");
		DATA_TYPE* A;
		DATA_TYPE* X;
		DATA_TYPE* Y;
		#ifdef HPC_DEBUG
			DATA_TYPE* host_A = new DATA_TYPE[NX * NY];
			DATA_TYPE* host_X = new DATA_TYPE[NY];
		#endif
		volatile DATA_TYPE* host_Y = new DATA_TYPE[NX];
		
		print_debug("[CUDA] Allocating A...");
		if(hipError_t err = hipMalloc((void**)&A, sizeof(DATA_TYPE) * NX * NY)) 
		{
			print_cudaError(err, "[CUDA] Could not allocate A!");
			return 1;
		}
		print_debug("[CUDA] Allocated A!");
		
		print_debug("[CUDA] Allocating X...");
		if(hipError_t err = hipMalloc((void**)&X, sizeof(DATA_TYPE) * NY))
		{
			print_cudaError(err, "[CUDA] Could not allocate X!");
			return 1;
		}
		print_debug("[CUDA] Allocated X!");

		print_debug("[CUDA] Allocating Y...");
		if(hipError_t err = hipMalloc((void**)&Y, sizeof(DATA_TYPE) * NX))
		{
			print_cudaError(err, "[CUDA] Could not allocate Y!");
			return 1;
		}
		print_debug("[CUDA] Allocated Y!");

		#ifdef HPC_INCLUDE_INIT
			print_debug("[Benchmark] Starting...");
			polybench_start_instruments;
		#endif

		print_debug("[Init] Initializing...");
		init_array_cuda<<<32, 32>>>((DATA_TYPE*) A, (DATA_TYPE*) X, (DATA_TYPE*) Y);
		if(hipError_t err = hipGetLastError())
		{
			print_cudaError(err, "[Init] Failed to execute kernel!");
			return 1;
		}
		print_debug("[Init] Complete!");

		#ifndef HPC_INCLUDE_INIT
			print_debug("[Benchmark] Starting...");
			polybench_start_instruments;
		#endif

		print_debug("[Kernel] Running...");
		kernel_atax_cuda<<<32, 32>>>((DATA_TYPE*) A, (DATA_TYPE*) X, (DATA_TYPE*) Y);
		print_debug("[Kernel] Complete!");

		#ifdef HPC_DEBUG
			print_debug("[CUDA] Copying A back...");
			if(hipError_t err = hipMemcpy(host_A, A, sizeof(DATA_TYPE) * NX * NY, hipMemcpyDeviceToHost)) {
				print_cudaError(err, "[CUDA] Could copy A back!");
				return 1;
			};
			print_debug("[CUDA] Copied A back!");

			print_debug("[CUDA] Copying X back...");
			if(hipError_t err = hipMemcpy(host_X, X, sizeof(DATA_TYPE) * NY, hipMemcpyDeviceToHost)) {
				print_cudaError(err, "[CUDA] Could copy X back!");
				return 1;
			};
			print_debug("[CUDA] Copied X back!");
		#endif

		print_debug("[CUDA] Copying Y back...");
		if(hipError_t err = hipMemcpy((void*) host_Y, Y, sizeof(DATA_TYPE) * NX, hipMemcpyDeviceToHost)) {
			print_cudaError(err, "[CUDA] Could copy Y back!");
			return 1;
		};
		print_debug("[CUDA] Copied Y back!");

		print_debug("[Benchmark] Stopping...");
		polybench_stop_instruments;
		polybench_print_instruments;
		print_debug("[Benchmark] Complete!");

		print_debug("[CUDA] Freeing A...");
		if(hipError_t err = hipFree(A)) {
			print_cudaError(err, "[CUDA] Could not free A!");
			return 1;
		}
		print_debug("[CUDA] Freed A!");

		print_debug("[CUDA] Freeing X...");
		if(hipError_t err = hipFree(X)) {
			print_cudaError(err, "[CUDA] Could not free X!");
			return 1;
		}
		print_debug("[CUDA] Freed X!");

		print_debug("[CUDA] Freeing Y...");
		if(hipError_t err = hipFree(Y)) {
			print_cudaError(err, "[CUDA] Could not free Y!");
			return 1;
		}
		print_debug("[CUDA] Freed Y!");

		#ifdef HPC_DEBUG
			print_debug("[Debug] Displaying A:");
			print_array(host_A, NX * NY);
			print_debug("[Debug] Displaying X:");
			print_array(host_X, NY);
			print_debug("[Debug] Displaying Y:");
			print_array((double*) host_Y, NX);
		#endif
	#endif

	return 0;
}
