#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <iostream>

/* Include polybench common header. */
#include "polybench.hu"

/* Include benchmark-specific header. */
/* Default data type is double, default size is 4000. */
#include "atax.hu"

// Workaround for the editor not finding M_PI
// It is exclusive to the GNU C compiler
// https://www.gnu.org/software/libc/manual/html_node/Mathematical-Constants.html
#ifndef M_PI
	#define M_PI 3.141
#endif

// Default if CUDA_NTHREADS is not set
#ifndef CUDA_NTHREADS
	#define CUDA_NTHREADS 128
#endif

// Enable syntax highlighting for the CUDA mode
// TODO: Remove this, as it will be set by .bench.sh
#define HPC_USE_CUDA

// Enable syntax highlighting for the stride mode
// TODO: Remove this, as it will be set by .bench.sh
#define HPC_USE_STRIDE

// Create macro for debug logging
#define debug(txt) std::cerr << txt << std::endl


/**
 * Initialize the arrays to be used in the computation:
 * 
 * - `X` is filled with multiples of `M_PI`;
 * - `Y` is zeroed;
 * - `A` is filled with sample data.
 * 
 * To be called on the CPU (uses the `__host__` qualifier).
 */
#ifndef HPC_USE_CUDA
__host__ static void init_array(DATA_TYPE** A, DATA_TYPE* X, DATA_TYPE* Y)
{
	/* X = [ 3.14, 6.28, 9.42, ... ] */
	for (unsigned int y = 0; y < NY; y++) 
	{
		X[y] = y * M_PI;
	}

	/* Y = [ 0.00, 0.00, 0.00, ... ] */
	for (unsigned int x = 0; x < NY; x++) 
	{
		Y[x] = 0;
	}

	/*
	 *	A = [
	 *	  [       0,       0,       0,       0, ... ],
	 *	  [  1 / NX,  2 / NX,  3 / NX,  4 / NX, ... ],
	 *	  [  2 / NX,  4 / NX,  6 / NX,  8 / NX, ... ],
	 *	  [  3 / NX,  6 / NX,  9 / NX, 12 / NX, ... ],
	 *	  ...
	 *	]
	 */
	for (unsigned int x = 0; x < NX; x++) 
	{
		for (unsigned int y = 0; y < NY; y++) 
		{
			A[x][y] = (DATA_TYPE)(x * (y + 1)) / NX;
		}
	}
}
#endif

/**
 * Initialize the `X` array.
 * 
 * Runs on the device.
 */
#ifdef HPC_USE_CUDA
__device__ static void init_array_cuda_x(DATA_TYPE* X, unsigned int threads)
{
	// Find how many iterations should be performed by each thread
	unsigned int perThread = NY / threads;

	// Find the index of the current thread, even if threads span multiple blocks
	int blockThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;
	
	// Have each thread perform the previously determined number of iterations
	for(int stride = 0; stride < perThread; stride++) {
		// Find the index of the current iteration
		// This is equal to `y` of the init_array function
		int iterationIdx = blockThreadIdx * stride;

		// Prevent the thread from accessing unallocated memory
		if(iterationIdx < NY) {

			// Set the array element
			X[iterationIdx] = iterationIdx * M_PI;
		}
	}
}
#endif

/**
 * Initialize the `Y` array.
 * 
 * Runs on the device.
 */
#ifdef HPC_USE_CUDA
__device__ static void init_array_cuda_y(DATA_TYPE* Y, unsigned int threads)
{
	// Find how many iterations should be performed by each thread
	unsigned int perThread = NX / threads;

	// Find the index of the current thread, even if threads span multiple blocks
	int blockThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;
	
	// Have each thread perform the previously determined number of iterations
	for(int stride = 0; stride < perThread; stride++) {
		// Find the index of the current iteration
		// This is equal to `y` of the init_array function
		int iterationIdx = blockThreadIdx * stride;

		// Prevent the thread from accessing unallocated memory
		if(iterationIdx < NX) {

			// Set the array element
			Y[iterationIdx] = 0;
		}
	}
}
#endif

/**
 * Initialize the `A` array.
 * 
 * Runs on the device.
*/
#ifdef HPC_USE_CUDA
__device__ static void init_array_cuda_a(DATA_TYPE* A, unsigned int threads)
{
	// Find how many elements should be written in total
	unsigned int elements = NX * NY;

	// Find how many iterations should be performed by each thread
	unsigned int perThread = elements / threads;

	// Find the index of the current thread, even if threads span multiple blocks
	int blockThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;

	/* TODO */
}
#endif

/**
 * Initialize the arrays to be used in the computation:
 * 
 * - `X` is filled with multiples of `M_PI`;
 * - `Y` is zeroed;
 * - `A` is filled with sample data.
 * 
 * Beware that `A` here is a simple array, it is not a matrix, so elements are accessed via [y * NX + x] (I think?).
 * 
 * It is called by the host, runs on the device, and calls the other init_arrays on the device.
 */
#ifdef HPC_USE_CUDA
__global__ static void init_array_cuda(DATA_TYPE* A, DATA_TYPE* X, DATA_TYPE* Y)
{
	unsigned int threads = gridDim.x * blockDim.x;

	init_array_cuda_x(X, threads);
	init_array_cuda_y(Y, threads);
	init_array_cuda_a(A, threads);
}
#endif

/**
 * Print the given array.
 * 
 * Cannot be parallelized, as the elements of the array should be 
 * 
 * To be called on the CPU (uses the `__host__` qualifier).
 */
__host__ static void print_array(DATA_TYPE* Y)
{
	for (unsigned int x = 0; x < NX; x++) 
	{
		fprintf(stderr, DATA_PRINTF_MODIFIER, Y[x]);
	}
	fprintf(stderr, "\n");
}


/**
 * Compute ATAX :
 * - A is the input matrix
 * - X is an input vector
 * - Y is the result vector
 * 
 * In particular:
 * ```
 * A * (A * X) = Y
 * ```
 * Wait, there's no transposition here?!?
 * 
 * Parallelizing this is the goal of the assignment.
 * 
 * Currently to be called on the CPU (uses the `__host__` qualifier), but we may probably want to change that soon.
 */
__host__ static void kernel_atax(DATA_TYPE** A, DATA_TYPE* X, DATA_TYPE* Y)
{
	for (unsigned int x = 0; x < NX; x++) 
	{
		DATA_TYPE tmp = 0;
		
		for (unsigned int y = 0; y < NY; y++) 
		{
			tmp += A[x][y] * X[y];
		}
		
		for (unsigned int y = 0; y < NY; y++) 
		{
			Y[y] += A[x][y] * tmp;
		}
	}
}


/**
 * The main function of the benchmark, which sets up tooling to measure the time spent computing `kernel_atax`.
 * 
 * We should probably avoid editing this.
 */
__host__ int main(int argc, char** argv)
{
	debug("Starting main...");

	#ifndef HPC_USE_CUDA

		debug("[Mode] Host-only");

		debug("[Pointers] Allocating...");

		// A[NX][NY]
		DATA_TYPE** A = new DATA_TYPE*[NX] {};
		for(unsigned int x = 0; x < NX; x++)
		{
			A[x] = new DATA_TYPE[NY] {};
		}

		// X[NY]
		DATA_TYPE* X = new DATA_TYPE[NY] {};

		// Y[NX]
		DATA_TYPE* Y = new DATA_TYPE[NX] {};

		debug("[Pointers] Allocated!");

		#ifdef HPC_INCLUDE_INIT
			debug("[Benchmark] Starting...");
			polybench_start_instruments;
		#endif

		debug("[Init] Initializing...");
		init_array(A, X, Y);
		debug("[Init] Initialized!");

		#ifndef HPC_INCLUDE_INIT
			debug("[Benchmark] Starting...");
			polybench_start_instruments;
		#endif

		debug("[Kernel] Running...");
		kernel_atax(A, X, Y);
		debug("[Kernel] Completed!");

		debug("[Benchmark] Stopping...");
		polybench_stop_instruments;
		polybench_print_instruments;
		debug("[Benchmark] Complete!");

		debug("[Verify] Printing...")
		polybench_prevent_dce(
			print_array(Y)
		);
		debug("[Verify] Done!")

	#else
	
		debug("[Mode] Host-and-device, CUDA");

		debug("[Pointers] Allocating...");
		DATA_TYPE* A;
		DATA_TYPE* X;
		DATA_TYPE* Y;
		
		debug("[CUDA] Allocating A...");
		if(hipMalloc((void**)&A, sizeof(DATA_TYPE) * NX * NY)) 
		{
			debug("[CUDA] Could not allocate A!");
			return 1;
		}
		debug("[CUDA] Allocated A!");
		
		debug("[CUDA] Allocating X...");
		if(hipMalloc((void**)&X, sizeof(DATA_TYPE) * NY))
		{
			debug("[CUDA] Could not allocate X!");
			return 1;
		}
		debug("[CUDA] Allocated X!");

		debug("[CUDA] Allocating Y...");
		if(hipMalloc((void**)&Y, sizeof(DATA_TYPE) * NX))
		{
			debug("[CUDA] Could not allocate Y!");
			return 1;
		}
		debug("[CUDA] Allocated Y!");

		#ifdef POLYBENCH_INCLUDE_INIT
			debug("[Benchmark] Starting...");
			polybench_start_instruments;
		#endif

		debug("[Init] Initializing...");
		init_array_cuda<<<32, 32>>>((double*) A, (double*) X, (double*) Y);
		if(hipGetLastError())
		{
			debug("[Init] Failed to execute kernel!");
			return 1;
		}
		debug("[Init] Initialized!");

		#ifndef POLYBENCH_INCLUDE_INIT
			debug("[Benchmark] Starting...");
			polybench_start_instruments;
		#endif

		// kernel_atax_cuda<<<1, 1>>>();

		polybench_stop_instruments;
		polybench_print_instruments;

		// Y = hipMemcpy();

		/*
		polybench_prevent_dce(
			print_array(Y)
		);
		*/

	#endif

	return 0;
}
