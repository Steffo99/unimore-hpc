#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <math.h>
#include <iostream>

/* Include polybench common header. */
#include "polybench.hu"

/* Include benchmark-specific header. */
/* Default data type is double, default size is 4000. */
#include "atax.hu"

// Workaround for the editor not finding M_PI
// It is exclusive to the GNU C compiler
// https://www.gnu.org/software/libc/manual/html_node/Mathematical-Constants.html
#ifndef M_PI
	#define M_PI 3.141
#endif

// Default if CUDA_NTHREADS is not set
#ifndef CUDA_NTHREADS
	#define CUDA_NTHREADS 128
#endif

// Enable syntax highlighting for the CUDA mode
// TODO: Remove this, as it will be set by .bench.sh
#define HPC_USE_CUDA

// Enable syntax highlighting for the stride mode
// TODO: Remove this, as it will be set by .bench.sh
#define HPC_USE_STRIDE


/**
 * Initialize the arrays to be used in the computation:
 * 
 * - `X` is filled with multiples of `M_PI`;
 * - `Y` is zeroed;
 * - `A` is filled with sample data.
 * 
 * To be called on the CPU (uses the `__host__` qualifier).
 */
#ifndef HPC_USE_CUDA
__host__ static void init_array(DATA_TYPE** A, DATA_TYPE* X, DATA_TYPE* Y)
{
	/* X = [ 3.14, 6.28, 9.42, ... ] */
	for (unsigned int y = 0; y < NY; y++) 
	{
		X[y] = y * M_PI;
	}

	/* Y = [ 0.00, 0.00, 0.00, ... ] */
	for (unsigned int x = 0; x < NY; x++) 
	{
		Y[x] = 0;
	}

	/*
	 *	A = [
	 *	  [       0,       0,       0,       0, ... ],
	 *	  [  1 / NX,  2 / NX,  3 / NX,  4 / NX, ... ],
	 *	  [  2 / NX,  4 / NX,  6 / NX,  8 / NX, ... ],
	 *	  [  3 / NX,  6 / NX,  9 / NX, 12 / NX, ... ],
	 *	  ...
	 *	]
	 */
	for (unsigned int x = 0; x < NX; x++) 
	{
		for (unsigned int y = 0; y < NY; y++) 
		{
			A[x][y] = (DATA_TYPE)(x * (y + 1)) / NX;
		}
	}
}
#endif

/**
 * Initialize the arrays to be used in the computation:
 * 
 * - `X` is filled with multiples of `M_PI`;
 * - `Y` is zeroed;
 * - `A` is filled with sample data.
 * 
 * It is called by the host, runs on the device, and calls the other init_arrays on the device.
 */
#ifdef HPC_USE_CUDA
__global__ static void init_array_cuda(DATA_TYPE** A, DATA_TYPE* X, DATA_TYPE* Y)
{
	unsigned int threads = gridDim.x * blockDim.x;

	init_array_cuda_x(X, threads);
	init_array_cuda_y(Y, threads);
	init_array_cuda_a(A, threads);
}
#endif

/**
 * Initialize the `X` array.
 * 
 * Runs on the device.
 */
#ifdef HPC_USE_CUDA
__device__ static void init_array_cuda_x(DATA_TYPE* X, unsigned int threads)
{
	// Find how many iterations should be performed by each thread
	unsigned int perThread = NY / threads;

	// Find the index of the current thread, even if threads span multiple blocks
	int blockThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;
	
	// Have each thread perform the previously determined number of iterations
	for(int stride = 0; stride < perThread; stride++) {
		// Find the index of the current iteration
		// This is equal to `y` of the init_array function
		int iterationIdx = blockThreadIdx * stride;

		// Prevent the thread from accessing unallocated memory
		if(iterationIdx < NY) {

			// Set the array element
			X[iterationIdx] = iterationIdx * M_PI;
		}
	}
}
#endif

/**
 * Initialize the `Y` array.
 * 
 * Runs on the device.
 */
#ifdef HPC_USE_CUDA
__device__ static void init_array_cuda_y(DATA_TYPE* Y, unsigned int threads)
{
	// Find how many iterations should be performed by each thread
	unsigned int perThread = NX / threads;

	// Find the index of the current thread, even if threads span multiple blocks
	int blockThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;
	
	// Have each thread perform the previously determined number of iterations
	for(int stride = 0; stride < perThread; stride++) {
		// Find the index of the current iteration
		// This is equal to `y` of the init_array function
		int iterationIdx = blockThreadIdx * stride;

		// Prevent the thread from accessing unallocated memory
		if(iterationIdx < NX) {

			// Set the array element
			Y[iterationIdx] = 0;
		}
	}
}
#endif

/**
 * Initialize the `A` array.
 * 
 * Runs on the device.
*/
#ifdef HPC_USE_CUDA
__device__ static void init_array_cuda_a(DATA_TYPE** A, unsigned int threads)
{

}
#endif

/**
 * Print the given array.
 * 
 * Cannot be parallelized, as the elements of the array should be 
 * 
 * To be called on the CPU (uses the `__host__` qualifier).
 */
__host__ static void print_array(DATA_TYPE* Y)
{
	for (unsigned int x = 0; x < NX; x++) 
	{
		fprintf(stderr, DATA_PRINTF_MODIFIER, Y[x]);
	}
	fprintf(stderr, "\n");
}


/**
 * Compute ATAX :
 * - A is the input matrix
 * - X is an input vector
 * - Y is the result vector
 * 
 * In particular:
 * ```
 * A * (A * X) = Y
 * ```
 * Wait, there's no transposition here?!?
 * 
 * Parallelizing this is the goal of the assignment.
 * 
 * Currently to be called on the CPU (uses the `__host__` qualifier), but we may probably want to change that soon.
 */
__host__ static void kernel_atax(DATA_TYPE** A, DATA_TYPE* X, DATA_TYPE* Y)
{
	for (unsigned int x = 0; x < NX; x++) 
	{
		DATA_TYPE tmp = 0;
		
		for (unsigned int y = 0; y < NY; y++) 
		{
			tmp += A[x][y] * X[y];
		}
		
		for (unsigned int y = 0; y < NY; y++) 
		{
			Y[y] += A[x][y] * tmp;
		}
	}
}


/**
 * The main function of the benchmark, which sets up tooling to measure the time spent computing `kernel_atax`.
 * 
 * We should probably avoid editing this.
 */
__host__ int main(int argc, char** argv)
{
	#ifndef HPC_USE_CUDA

		// A[NX][NY]
		DATA_TYPE** A = new DATA_TYPE*[NX] {};
		for(unsigned int x = 0; x < NX; x++)
		{
			A[x] = new DATA_TYPE[NY] {};
		}

		// X[NY]
		DATA_TYPE* X = new DATA_TYPE[NY] {};

		// Y[NX]
		DATA_TYPE* Y = new DATA_TYPE[NX] {};

		#ifdef HPC_INCLUDE_INIT
			polybench_start_instruments;
		#endif

		init_array(A, X, Y);

		#ifndef HPC_INCLUDE_INIT
			polybench_start_instruments;
		#endif

		kernel_atax(A, X, Y);

		polybench_stop_instruments;
		polybench_print_instruments;

		polybench_prevent_dce(
			print_array(Y)
		);

	#else

		DATA_TYPE** A;
		DATA_TYPE* X;
		DATA_TYPE* Y;
		
		if(hipMalloc(&A, sizeof(DATA_TYPE) * NX * NY)) 
		{
			std::cerr << "Could not allocate A on the device\n";
			return 1;
		}
		
		if(hipMalloc(&X, sizeof(DATA_TYPE) * NY))
		{
			std::cerr << "Could not allocate X on the device\n";
			return 1;
		}

		if(hipMalloc(&Y, sizeof(DATA_TYPE) * NX))
		{
			std::cerr << "Could not allocate Y on the device\n";
			return 1;
		}

		#ifdef POLYBENCH_INCLUDE_INIT
			polybench_start_instruments;
		#endif

		init_array_cuda<<<1, 1>>>(A, X, Y);

		#ifndef POLYBENCH_INCLUDE_INIT
			polybench_start_instruments;
		#endif

		// kernel_atax_cuda<<<1, 1>>>();

		polybench_stop_instruments;
		polybench_print_instruments;

		// Y = hipMemcpy();

		/*
		polybench_prevent_dce(
			print_array(Y)
		);
		*/

	#endif

	return 0;
}
